#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <errno.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

#pragma pack(push,1)
typedef struct {
  char         filetype[2];   /* magic - always 'B' 'M' */
  unsigned int filesize;
  short        reserved1;
  short        reserved2;
  unsigned int dataoffset;    /* offset in bytes to actual bitmap data */
} file_header;

typedef struct {
  file_header  fileheader;
  unsigned int headersize;
  int          width;
  int          height;
  short        planes;
  short        bitsperpixel;  /* we only support the value 24 here */
  unsigned int compression;   /* we do not support compression */
  unsigned int bitmapsize;
  int          horizontalres;
  int          verticalres;
  unsigned int numcolors;
  unsigned int importantcolors;
} bitmap_header;
#pragma pack(pop)

typedef struct _bitmap_metadata {
  int width;
  int height;
  int threads;
  int blurSize;
} bitmap_metadata;

__global__ void d_blur(unsigned char* d_data, bitmap_metadata* d_bitmap_metadata) {
  int thread = (blockDim.x * blockIdx.x) + threadIdx.x;
  int x, y, xx, yy, avgB, avgG, avgR, ile;
  int xmin = d_bitmap_metadata->width / d_bitmap_metadata->threads * thread;
  int xmax = d_bitmap_metadata->width / d_bitmap_metadata->threads * (thread + 1);
  int ymin = 0;
  int ymax = d_bitmap_metadata->height;

  if(thread < d_bitmap_metadata->width)
    for(xx = xmin; xx < xmax; xx++) {
      for(yy = ymin; yy < ymax; yy++) {
        avgB = avgG = avgR = ile = 0;

        for(x = xx; x < d_bitmap_metadata->width && x < xx + d_bitmap_metadata->blurSize; x++) {
          for(y = yy; y < d_bitmap_metadata->height && y < yy + d_bitmap_metadata->blurSize; y++) {
            avgB += d_data[x * 3 + y * d_bitmap_metadata->width * 3 + 0];
            avgG += d_data[x * 3 + y * d_bitmap_metadata->width * 3 + 1];
            avgR += d_data[x * 3 + y * d_bitmap_metadata->width * 3 + 2];
            ile++;
          }
        }

        avgB /= ile;
        avgG /= ile;
        avgR /= ile;

        d_data[xx * 3 + yy * d_bitmap_metadata->width * 3 + 0] = avgB;
        d_data[xx * 3 + yy * d_bitmap_metadata->width * 3 + 1] = avgG;
        d_data[xx * 3 + yy * d_bitmap_metadata->width * 3 + 2] = avgR;
      }
    }
}

int blur(char* input, char *output, int kernel) {
  int dev = 0;
  hipError_t error = hipSuccess;
  hipSetDevice(dev);
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, dev);

  int threadsPerBlock, threads;
  int blocksPerGrid = deviceProp.multiProcessorCount;

  FILE *fp, *out;
  bitmap_header* hp;
  int n, blurSize = kernel;
  unsigned char *data, *d_data;
  bitmap_metadata *d_bitmap_metadata, *h_bitmap_metadata;

  fp = fopen(input, "r");

  hp = (bitmap_header*) malloc(sizeof(bitmap_header));
  if(hp == NULL)
    return 3;

  n = fread(hp, sizeof(bitmap_header), 1, fp);
  if(n != 1)
    printf("Error reading file\n");
  data = (unsigned char*) malloc(sizeof(unsigned char) * hp->bitmapsize);

  fseek(fp, sizeof(char) * hp->fileheader.dataoffset, SEEK_SET);
  n = fread(data, sizeof(char), hp->bitmapsize, fp);

  h_bitmap_metadata = (bitmap_metadata*) malloc(sizeof(bitmap_metadata));

  threads = hp->width;
  threadsPerBlock = threads / blocksPerGrid;

  h_bitmap_metadata->width = hp->width;
  h_bitmap_metadata->height = hp->height;
  h_bitmap_metadata->threads = threads;
  h_bitmap_metadata->blurSize = blurSize;

  error = hipMalloc(&d_data, sizeof(unsigned char) * hp->bitmapsize);
  if(error != hipSuccess) printf("Error alocating memory\n");
  error = hipMalloc(&d_bitmap_metadata, sizeof(bitmap_metadata));
  if(error != hipSuccess) printf("Error alocating memory\n");

  error = hipMemcpy(d_data, data, sizeof(unsigned char) * hp->bitmapsize, hipMemcpyHostToDevice);
  if(error != hipSuccess) printf("Error transferring data from host to device\n");
  error = hipMemcpy(d_bitmap_metadata, h_bitmap_metadata, sizeof(bitmap_metadata), hipMemcpyHostToDevice);
  if(error != hipSuccess) printf("Error transferring data from host to device\n");

  d_blur<<<blocksPerGrid, threadsPerBlock>>>(d_data, d_bitmap_metadata);
  error = hipGetLastError();
  if(error != hipSuccess) {
    printf("Kernel d_blur function error: %s\n", hipGetErrorString(error));
  }

  error = hipMemcpy(data, d_data, sizeof(unsigned char) * hp->bitmapsize, hipMemcpyDeviceToHost);
  if(error != hipSuccess) {
    printf("Error transferring data from device to host\n");
    printf("Error: %s\n", hipGetErrorString(error));
  }

  out = fopen(output, "wb");

  n = fwrite(hp, sizeof(char), sizeof(bitmap_header), out);
  fseek(out, sizeof(char) * hp->fileheader.dataoffset, SEEK_SET);
  n = fwrite(data, sizeof(char), hp->bitmapsize, out);

  fclose(fp);
  fclose(out);
  free(hp);
  free(data);
  hipFree(d_data);
  hipFree(d_bitmap_metadata);
  return 0;
}

double time_spec_seconds(struct timespec* ts) {
  return (double) ts->tv_sec + (double) ts->tv_nsec * 1.0e-9;
}

int main(int argc, char **argv) {
  struct timespec tstart = {0,0}, tend = {0,0};
  char* original = argv[1];
  char* modified = argv[2];
  int kernel =  atoi(argv[3]);

  if (kernel < 3 || kernel > 15)
    printf("Invalid kernel value. Must be between [3, 15]");
  else {
    int r = clock_gettime(CLOCK_MONOTONIC, &tstart);

    if (r == -1) {
      printf("The clock_gettime() function failed: %s\n", strerror(errno));
      return 1;
    }

    blur(original, modified, kernel);

    r = clock_gettime(CLOCK_MONOTONIC, &tend);
    if (r == -1) {
      printf("The clock_gettime() function failed: %s\n", strerror(errno));
      return 1;
    }

    double delta = time_spec_seconds(&tend) - time_spec_seconds(&tstart);

    printf("%d\t%.4f\n", kernel, delta);
  }
}
